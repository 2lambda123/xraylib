#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "xrayglob.h"


#ifdef __cplusplus
#define XRLCUDACPLUSPLUS extern "C"
#else
#define XRLCUDACPLUSPLUS
#endif



#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}


__device__ float *FluorYield_arr_d;



XRLCUDACPLUSPLUS int CudaXRayInit() {



	int deviceCount, device;
	int gpuDeviceCount = 0;
	hipDeviceProp_t properties;
	hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
	if (cudaResultCode != hipSuccess) 
        	deviceCount = 0;
   	/* machines with no GPUs can still report one emulation device */
	for (device = 0; device < deviceCount; ++device) {
        	hipGetDeviceProperties(&properties, device);
        	if (properties.major != 9999) /* 9999 means emulation only */
            		++gpuDeviceCount;
    	}

    	/* don't just return the number of gpus, because other runtime cuda
       	errors can also yield non-zero return values */
    	if (gpuDeviceCount == 0) {
		fprintf(stderr,"No CUDA enabled devices found\nAborting\n");
        	return 0;
	}



	/* start malloc'ing and memcpy'ing */
     	CudaSafeCall(hipMalloc((void **)&FluorYield_arr_d,sizeof(float)*(ZMAX+1)*SHELLNUM));
  	CudaSafeCall(hipMemcpy(FluorYield_arr_d, FluorYield_arr, sizeof(float)*(ZMAX+1)*SHELLNUM, hipMemcpyHostToDevice));

	



	return 1;
}





