#include "hip/hip_runtime.h"
#include <stdio.h>
#include "xraylib.h"
#include "xraylib-cuda.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>

//#define CUDA_ERROR_CHECK


__global__ void Yields(int *Z, int *shells, float *yields) {
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	printf("tid: %i\n", tid);

	yields[tid] = FluorYield_cu(Z[tid], shells[tid]);
	//yields[tid] = (float) tid; 
	//printf("yield: %f\n",yields[tid]);
	
	__syncthreads();
	return;
}




int main (int argc, char *argv[]) {

	fprintf(stdout,"Entering xrlexample11\n");
	
	int Z[5] = {10,15,26,79,82};
	int shells[5] = {K_SHELL, K_SHELL, K_SHELL, L3_SHELL,L1_SHELL};
	int *Zd;
	int *shellsd;

	float yields[5], *yieldsd;

	CudaXRayInit();

	//fluorescence yields example
	CudaSafeCall(hipMalloc((void **) &Zd, 5*sizeof(int)));
	CudaSafeCall(hipMemcpy(Zd, Z, 5*sizeof(int), hipMemcpyHostToDevice));
	CudaSafeCall(hipMalloc((void **) &shellsd, 5*sizeof(int)));
	CudaSafeCall(hipMemcpy(shellsd, shells, 5*sizeof(int), hipMemcpyHostToDevice));

	CudaSafeCall(hipMalloc((void **) &yieldsd, 5*sizeof(float)));

	Yields<<<1,5>>>(Zd, shellsd,yieldsd);	

	CudaCheckError();

	
	CudaSafeCall(hipMemcpy(yields, yieldsd, 5*sizeof(float), hipMemcpyDeviceToHost));

	fprintf(stdout,"Fluorescence yields\n");
	fprintf(stdout,"Shell   Classic   CUDA\n");
	fprintf(stdout,"Ne-K    %8f %f\n",FluorYield(10,K_SHELL), yields[0]);
	fprintf(stdout,"P-K     %8f %f\n",FluorYield(15,K_SHELL), yields[1]);
	fprintf(stdout,"Fe-K    %8f %f\n",FluorYield(26,K_SHELL), yields[2]);
	fprintf(stdout,"Au-K    %8f %f\n",FluorYield(79,L3_SHELL), yields[3]);
	fprintf(stdout,"Pb-K    %8f %f\n",FluorYield(82,L1_SHELL), yields[4]);


}
